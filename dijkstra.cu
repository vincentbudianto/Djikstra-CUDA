// Copyright www.computing.llnl.gov

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <limits.h>
#include <omp.h>
#include <assert.h>
#include <time.h>

__device__ int minDistance(long dist[], bool sptSet[], long V)
{
    // Initialize min value
    int min = INT_MAX, min_index;

    for (int v = 0; v < V; v++)
    {
        if ((sptSet[v] == false) && (dist[v] <= min))
        {
            min = dist[v], min_index = v;
        }
    }

    return min_index;
}

__device__ void dijkstra(long src, long V, long *graph, long *dist)
{
    // sptSet[i] will be true if vertex i is included in shortest path tree or shortest distance from src to i is finalized
    bool *sptSet = (bool*)malloc(V);

    // Initialize all distances as INFINITE and stpSet[] as false
    for (int i = 0; i < V; i++)
    {
        dist[i] = INT_MAX, sptSet[i] = false;
    }

    // Distance of source vertex from itself is always 0
    dist[src] = 0;

    // Find shortest path for all vertices
    for (int count = 0; count < V - 1; count++)
    {
        // Pick the minimum distance vertex from the set of vertices not yet processed. u is always equal to src in the first iteration.
        int u = minDistance(dist, sptSet, V);

        // Mark the picked vertex as processed
        sptSet[u] = true;

        // Update dist value of the adjacent vertices of the picked vertex.
        for (int v = 0; v < V; v++)
        {
            // Update dist[v] only if is not in sptSet, there is an edge from u to v, and total weight of path from src to  v through u is smaller than current value of dist[v]
            if (!sptSet[v] && graph[(u * V) + v] && (dist[u] != INT_MAX) && (dist[u] + graph[(u * V) + v] < dist[v]))
            {
                dist[v] = dist[u] + graph[(u * V) + v];
            }
        }
    }
}

__global__ void solution(long *matrix, long *newmatrix, long nodes) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    long *dist = (long*)malloc(nodes);
    for (int i = index; i < nodes; i += stride)
    {
        dijkstra(i, nodes, matrix, dist);
        // printf("Completing part %d with processor %d\n", i, rank);
        // MPI_Send(newmatrix[i], nodes, MPI_INT, 0, i, MPI_COMM_WORLD);
        memcpy(&newmatrix[i*nodes], dist, nodes*sizeof(*dist));
    }

}

int main(int argc, char *argv[])
{
    // Set rand() seed
    srand(13517020);
    /* total cost == 0
     size | total
     100  |    0
     500  |    4
     1000 |   18
     3000 |  135
     5000 |  380
	*/
    // srand(13517137);
    /* total cost == 0
     size | total
     100  |    0
     500  |    2
     1000 |   13
     3000 |  126
     5000 |  360
  	*/

    if (argc < 2)
    {
        fprintf(stderr, "error: missing command line arguments\n");
        exit(1);
    }
    else
    {
        clock_t begin = clock();
        // Inititate graph
        long nodes = atoi(argv[1]);
        long num_bytes = nodes*nodes*sizeof(long);
        long *d_matrix, *h_matrix = 0;

        h_matrix = (long*)malloc(num_bytes);
        hipMalloc((void**)&d_matrix, num_bytes);

        if (0==h_matrix || 0==d_matrix) {
            printf("Couldn't allocate memory\n");
            return 1;
        }

        hipMemset(d_matrix,0,num_bytes);
        
        // Build graph
        for (int i = 0; i < nodes; i++)
        {
            for (int j = 0; j < nodes; j++)
            {
                if (i == j)
                {
                    h_matrix[(i*nodes) + j] = 0;
                }
                else
                {
                    h_matrix[(i*nodes) + j] = rand();
                }
            }
        }

        hipMemcpy(d_matrix,h_matrix,num_bytes,hipMemcpyHostToDevice);
        
        long *d_newMatrix, *h_newMatrix = 0;

        h_newMatrix = (long*)malloc(num_bytes);
        hipMalloc((void**)&d_newMatrix, num_bytes);

        if (0==h_newMatrix || 0==d_newMatrix) {
            printf("Couldn't allocate newmatrix memory\n");
            return 1;
        }

        hipMemset(d_newMatrix,0,num_bytes);

        int blockSize = 256;
        int numBlocks = (nodes + blockSize - 1) / blockSize;
        solution<<<numBlocks, blockSize>>>(d_matrix, d_newMatrix, nodes);

        hipMemcpy( h_newMatrix, d_newMatrix, num_bytes, hipMemcpyDeviceToHost );

        clock_t end = clock();
        // printf("Printing to file");
        // Write to file
        FILE *fp;
        fp = fopen("old_matrix.txt", "w");
        fprintf(fp, "Old matrix:\n");

        for (int i = 0; i < nodes; i++)
        {
            for (int j = 0; j < nodes; j++)
            {
                fprintf(fp, "%ld ", h_matrix[(i*nodes) +j]);
            }

            fprintf(fp, "\n");
        }

        fclose(fp);

        fp = fopen("result.txt", "w");
        fprintf(fp, "New matrix:\n");

        for (int i = 0; i < nodes; i++)
        {
            for (int j = 0; j < nodes; j++)
            {
                fprintf(fp, "%ld ", h_newMatrix[(i*nodes) + j]);
            }

            fprintf(fp, "\n");
        }

        fprintf(fp, "Solution found in: %.3f microseconds\n", ((double)(end - begin) / CLOCKS_PER_SEC) * 1000000);
        printf("Solution found in: %.3f microseconds\n", ((double)(end - begin) / CLOCKS_PER_SEC) * 1000000);
        fclose(fp);

        // Dealocation

        free(h_matrix);
        free(h_newMatrix);
        hipFree(d_matrix);
        hipFree(d_newMatrix);
    }
    return 0;
}